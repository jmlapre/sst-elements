// Copyright 2009-2023 NTESS. Under the terms
// of Contract DE-NA0003525 with NTESS, the U.S.
// Government retains certain rights in this software.
//
// Copyright (c) 2009-2023, NTESS
// All rights reserved.
//
// Portions are copyright of other developers:
// See the file CONTRIBUTORS.TXT in the top level directory
// of the distribution for more information.
//
// This file is part of the SST software package. For license
// information, see the LICENSE file in the top level directory of the
// distribution.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

int main()
{
    // Size of vectors
    int n = 16384;

    // Host input vectors
    double *h_a;
    double *h_b;
	// Host output vector
    double *h_c;

    // Device input vectors
    double *d_a;
    double *d_b;
    // Device output vector
    double *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(double);

    // Allocate memory for each vector on host
    h_a = (double *)malloc(bytes);
    h_b = (double *)malloc(bytes);
    h_c = (double *)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 256;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i = 0; i < n; i++)
        sum += h_c[i];
    printf("final result: %.5f\n", sum / n);
    fflush(stdout);
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
